#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 #include "BatchData.h"
 #include "zlib.h"
 #include "nvcomp/gzip.h"

// modelled after BatchData(BatchDataCPU, bool) and operator==(BatchDataCPU, BatchData)
// seems i didn't need to write this and could have just used BatchDataCPU(void**, size_t*, uint8_t*, size_t, bool)
BatchDataCPU::BatchDataCPU(const BatchData& batch_data, bool copy_data) :
  m_ptrs(),
  m_sizes(),
  m_data(),
  m_size()
{
  m_size = batch_data.size();
  m_sizes = std::vector<size_t>(batch_data.size());
  CUDA_CHECK(hipMemcpy(
      m_sizes.data(),
      batch_data.sizes(),
      batch_data.size() * sizeof(size_t),
      hipMemcpyDeviceToHost));

  size_t data_size = std::accumulate(
        sizes(),
        sizes() + size(),
        static_cast<size_t>(0));
  m_data = std::vector<uint8_t>(data_size);

  size_t offset = 0;
  std::vector<void*> ptrs(size());
  for (size_t i = 0; i < size(); ++i) {
    ptrs[i] = data() + offset;
    offset += sizes()[i];
  }
  m_ptrs = std::vector<void*>(ptrs);

  if (copy_data) {
    std::vector<void*> src(batch_data.size());
    CUDA_CHECK(hipMemcpy(
        src.data(),
        batch_data.ptrs(),
        batch_data.size() * sizeof(void*),
        hipMemcpyDeviceToHost));

    const size_t* bytes = sizes();
    for (size_t i = 0; i < size(); ++i) {
      CUDA_CHECK(
          hipMemcpy(ptrs[i], src[i], bytes[i], hipMemcpyDeviceToHost));
    }
  }
}

 // Benchmark performance from the binary data file fname
 static void run_example(const std::vector<std::vector<char>>& data)
 {
   size_t total_bytes = 0;
   std::vector<size_t> comp_sizes;
   std::vector<size_t> decomp_sizes;
   size_t max_decomp_size = 0;
   for (const std::vector<char>& part : data) {
     comp_sizes.push_back(part.size());
     // get uncompressed size of file from gzip footer
     size_t size = *(size_t*)(part.data() + part.size() - 4);
     total_bytes += size;
     decomp_sizes.push_back(size);
     max_decomp_size = std::max(max_decomp_size, size);
   }
 
   std::cout << "----------" << std::endl;
   std::cout << "files: " << data.size() << std::endl;
   std::cout << "uncompressed (B): " << total_bytes << std::endl;
 
   // seems to be the maximum chunk size
   const size_t chunk_size = 1 << 16;
 
   // build up input batch on CPU
   BatchDataCPU compress_data_cpu(data, comp_sizes);
   std::cout << "chunks: " << compress_data_cpu.size() << std::endl;
 
   // compute compression ratio
   size_t* compressed_sizes_host = compress_data_cpu.sizes();
   size_t comp_bytes = 0;
   for (size_t i = 0; i < compress_data_cpu.size(); ++i)
     comp_bytes += compressed_sizes_host[i];
 
   std::cout << "comp_size: " << comp_bytes
             << ", compressed ratio: " << std::fixed << std::setprecision(2)
             << (double)total_bytes / comp_bytes << std::endl;
 
   // Copy compressed data to GPU
   BatchData compress_data(compress_data_cpu, true);
 
   // Allocate and build up decompression batch on GPU
   BatchData decomp_data(total_bytes, decomp_sizes);
 
   // Create CUDA stream
   hipStream_t stream;
   hipStreamCreate(&stream);
 
   // CUDA events to measure decompression time
   hipEvent_t start, end;
   hipEventCreate(&start);
   hipEventCreate(&end);

   // deflate GPU decompression
   size_t decomp_temp_bytes;
   nvcompStatus_t status = nvcompBatchedGzipDecompressGetTempSize(
       compress_data.size(), max_decomp_size, &decomp_temp_bytes);
   if (status != nvcompSuccess) {
     throw std::runtime_error("nvcompBatchedGzipDecompressGetTempSize() failed.");
   }
 
   void* d_decomp_temp;
   CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));
 
   size_t* d_decomp_sizes;
   CUDA_CHECK(hipMalloc(&d_decomp_sizes, decomp_data.size() * sizeof(size_t)));
 
   nvcompStatus_t* d_status_ptrs;
   CUDA_CHECK(hipMalloc(&d_status_ptrs, decomp_data.size() * sizeof(nvcompStatus_t)));
 
   CUDA_CHECK(hipStreamSynchronize(stream));
 
   // Run decompression
   status = nvcompBatchedGzipDecompressAsync(
       compress_data.ptrs(),
       compress_data.sizes(),
       decomp_data.sizes(),
       d_decomp_sizes,
       compress_data.size(),
       d_decomp_temp,
       decomp_temp_bytes,
       decomp_data.ptrs(),
       d_status_ptrs,
       stream);
   if( status != nvcompSuccess){
     throw std::runtime_error("ERROR: nvcompBatchedGzipDecompressAsync() not successful");
   }

   BatchDataCPU decomp_data_cpu(decomp_data, true);
   std::cout << "chunks: " << decomp_data_cpu.size() << std::endl;

   // Re-run decompression to get throughput
   hipEventRecord(start, stream);
   status = nvcompBatchedGzipDecompressAsync(
     compress_data.ptrs(),
     compress_data.sizes(),
     decomp_data.sizes(),
     d_decomp_sizes,
     compress_data.size(),
     d_decomp_temp,
     decomp_temp_bytes,
     decomp_data.ptrs(),
     d_status_ptrs,
     stream);
   hipEventRecord(end, stream);
   if( status != nvcompSuccess){
     throw std::runtime_error("ERROR: nvcompBatchedGzipDecompressAsync() not successful");
   }
 
   CUDA_CHECK(hipStreamSynchronize(stream));
 
   float ms;
   hipEventElapsedTime(&ms, start, end);
 
   double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
   std::cout << "decompression throughput (GB/s): " << decompression_throughput
             << std::endl;
   std::cout << "ms: " << ms << std::endl;
 
   hipFree(d_decomp_temp);
 
   hipEventDestroy(start);
   hipEventDestroy(end);
   hipStreamDestroy(stream);
 }
 
 std::vector<char> readFile(const std::string& filename)
 {
   std::vector<char> buffer(4096);
   std::vector<char> host_data;
 
   std::ifstream fin(filename, std::ifstream::binary);
   fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);
 
   size_t num;
   do {
     num = fin.readsome(buffer.data(), buffer.size());
     host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
   } while (num > 0);
 
   return host_data;
 }
 
 std::vector<std::vector<char>>
 multi_file(const std::vector<std::string>& filenames)
 {
   std::vector<std::vector<char>> split_data;
 
   for (auto const& filename : filenames) {
     split_data.emplace_back(readFile(filename));
   }
 
   return split_data;
 }
 
 int main(int argc, char* argv[])
 {
    std::vector<std::string> file_names(argc - 1);

    if (argc == 1) {
      std::cerr << "Must specify at least one file." << std::endl;
      return 1;
    }
  
    // if `-f` is specified, assume single file mode
    if (strcmp(argv[1], "-f") == 0) {
      if (argc == 2) {
        std::cerr << "Missing file name following '-f'" << std::endl;
        return 1;
      } else if (argc > 3) {
        std::cerr << "Unknown extra arguments with '-f'." << std::endl;
        return 1;
      }
  
      file_names = {argv[2]};
    } else {
      // multi-file mode
      for (int i = 1; i < argc; ++i) {
        file_names[i - 1] = argv[i];
      }
    }
  
    auto data = multi_file(file_names);
  
    run_example(data);
 
   return 0;
 }
 