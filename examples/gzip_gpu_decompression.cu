/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 #include "BatchData.h"
 #include "zlib.h"
 #include "nvcomp/gzip.h"

// modelled after BatchData(BatchDataCPU, bool) and operator==(BatchDataCPU, BatchData)
// seems i didn't need to write this and could have just used BatchDataCPU(void**, size_t*, uint8_t*, size_t, bool)
BatchDataCPU::BatchDataCPU(const BatchData& batch_data, bool copy_data) :
  m_ptrs(),
  m_sizes(),
  m_data(),
  m_size()
{
  m_size = batch_data.size();std::cerr<<__LINE__<<std::endl;
  m_sizes = std::vector<size_t>(batch_data.size());std::cerr<<__LINE__<<std::endl;
  CUDA_CHECK(hipMemcpy(
      m_sizes.data(),
      batch_data.sizes(),
      batch_data.size() * sizeof(size_t),
      hipMemcpyDeviceToHost));std::cerr<<__LINE__<<std::endl;

  size_t data_size = std::accumulate(
        sizes(),
        sizes() + size(),
        static_cast<size_t>(0));std::cerr<<__LINE__<<std::endl;
        std::cerr << "data_size: " << data_size << std::endl;
  m_data = std::vector<uint8_t>(data_size);std::cerr<<__LINE__<<std::endl;

  size_t offset = 0;std::cerr<<__LINE__<<std::endl;
  std::vector<void*> ptrs(size());std::cerr<<__LINE__<<std::endl;
  for (size_t i = 0; i < size(); ++i) {
    ptrs[i] = data() + offset;
    offset += sizes()[i];
  }std::cerr<<__LINE__<<std::endl;
  m_ptrs = std::vector<void*>(ptrs);std::cerr<<__LINE__<<std::endl;

  if (copy_data) {
    std::vector<void*> src(batch_data.size());
    CUDA_CHECK(hipMemcpy(
        src.data(),
        batch_data.ptrs(),
        batch_data.size() * sizeof(void*),
        hipMemcpyDeviceToHost));

    const size_t* bytes = sizes();std::cerr<<__LINE__<<std::endl;
    for (size_t i = 0; i < size(); ++i) {
      CUDA_CHECK(
          hipMemcpy(ptrs[i], src[i], bytes[i], hipMemcpyDeviceToHost));
      // std::cerr << "i: " << i << ", ptrs[i][0-10]: " << ((char*)ptrs[i])[0] << ((char*)ptrs[i])[1] << ((char*)ptrs[i])[2] << ((char*)ptrs[i])[3] << ((char*)ptrs[i])[4] << ((char*)ptrs[i])[5] << ((char*)ptrs[i])[6] << ((char*)ptrs[i])[7] << ((char*)ptrs[i])[8] << ((char*)ptrs[i])[9] << ((char*)ptrs[i])[10] << " ptrs[i][-10:]: " << ((char*)ptrs[i])[bytes[i]-10] << ((char*)ptrs[i])[bytes[i]-9] << ((char*)ptrs[i])[bytes[i]-8] << ((char*)ptrs[i])[bytes[i]-7] << ((char*)ptrs[i])[bytes[i]-6] << ((char*)ptrs[i])[bytes[i]-5] << ((char*)ptrs[i])[bytes[i]-4] << ((char*)ptrs[i])[bytes[i]-3] << ((char*)ptrs[i])[bytes[i]-2] << ((char*)ptrs[i])[bytes[i]-1] << std::endl;
    }

    std::cerr << "strlen(ptrs[0]): " << strlen((char*)ptrs[0]) << std::endl;
  }
}

 // Benchmark performance from the binary data file fname
 static void run_example(const std::vector<std::vector<char>>& data)
 {
   size_t total_bytes = 0;
   std::vector<size_t> block_sizes;
   for (const std::vector<char>& part : data) {
     // get uncompressed size of file from gzip footer
     total_bytes += *(size_t*)(part.data() + part.size() - 4);
   }
 
   std::cout << "----------" << std::endl;
   std::cout << "files: " << data.size() << std::endl;
   std::cout << "uncompressed (B): " << total_bytes << std::endl;
 
   // seems to be the maximum chunk size
   const size_t chunk_size = 1 << 16;
 
   // build up input batch on CPU
   BatchDataCPU compress_data_cpu(data, data[0].size());
   std::cout << "chunks: " << compress_data_cpu.size() << std::endl;
 
   // compute compression ratio
   size_t* compressed_sizes_host = compress_data_cpu.sizes();
   size_t comp_bytes = 0;
   for (size_t i = 0; i < compress_data_cpu.size(); ++i)
     comp_bytes += compressed_sizes_host[i];
 
   std::cout << "comp_size: " << comp_bytes
             << ", compressed ratio: " << std::fixed << std::setprecision(2)
             << (double)total_bytes / comp_bytes << std::endl;
 
   // Copy compressed data to GPU
   BatchData compress_data(compress_data_cpu, true);
 
   // Allocate and build up decompression batch on GPU
   BatchData decomp_data(total_bytes, 1);
   std::cerr << "decomp_data.size(): " << decomp_data.size() << std::endl;
 
   // Create CUDA stream
   hipStream_t stream;
   hipStreamCreate(&stream);
 
   // CUDA events to measure decompression time
   hipEvent_t start, end;
   hipEventCreate(&start);
   hipEventCreate(&end);

   // deflate GPU decompression
   size_t decomp_temp_bytes;
   nvcompStatus_t status = nvcompBatchedGzipDecompressGetTempSize(
       compress_data.size(), chunk_size * 10, &decomp_temp_bytes);
   if (status != nvcompSuccess) {
     throw std::runtime_error("nvcompBatchedGzipDecompressGetTempSize() failed.");
   }
   std::cerr << "decomp_temp_bytes: " << decomp_temp_bytes << std::endl;
 
   void* d_decomp_temp;
   CUDA_CHECK(hipMalloc(&d_decomp_temp, decomp_temp_bytes));
 
   size_t* d_decomp_sizes;
   CUDA_CHECK(hipMalloc(&d_decomp_sizes, decomp_data.size() * sizeof(size_t)));
   std::cerr << "d_decomp_sizes: " << d_decomp_sizes << std::endl;
 
   nvcompStatus_t* d_status_ptrs;
   CUDA_CHECK(hipMalloc(&d_status_ptrs, decomp_data.size() * sizeof(nvcompStatus_t)));
 
   CUDA_CHECK(hipStreamSynchronize(stream));
 
   // Run decompression
   status = nvcompBatchedGzipDecompressAsync(
       compress_data.ptrs(),
       compress_data.sizes(),
       decomp_data.sizes(),
       d_decomp_sizes,
       compress_data.size(),
       d_decomp_temp,
       decomp_temp_bytes,
       decomp_data.ptrs(),
       d_status_ptrs,
       stream);
   if( status != nvcompSuccess){
     throw std::runtime_error("ERROR: nvcompBatchedGzipDecompressAsync() not successful");
   }

   size_t* decomp_sizes_host = (size_t*)malloc(decomp_data.size() * sizeof(size_t));
   std::cerr << "d_decomp_sizes: ";
    CUDA_CHECK(hipMemcpy(
        decomp_sizes_host,
        d_decomp_sizes,
        decomp_data.size() * sizeof(size_t),
        hipMemcpyDeviceToHost));
    for (size_t i = 0; i < decomp_data.size(); ++i) {
      std::cerr << decomp_sizes_host[i] << " ";
    }
    std::cerr << std::endl;
    free(decomp_sizes_host);

    std::cerr << "d_status_ptrs: ";
    nvcompStatus_t* status_ptrs_host = (nvcompStatus_t*)malloc(decomp_data.size() * sizeof(nvcompStatus_t));
    CUDA_CHECK(hipMemcpy(
        status_ptrs_host,
        d_status_ptrs,
        decomp_data.size() * sizeof(nvcompStatus_t),
        hipMemcpyDeviceToHost));
    for (size_t i = 0; i < decomp_data.size(); ++i) {
      std::cerr << status_ptrs_host[i] << " ";
    }
    std::cerr << std::endl;
    free(status_ptrs_host);
 
   BatchDataCPU decomp_data_cpu(decomp_data, true);
   std::cout << "chunks: " << decomp_data_cpu.size() << std::endl;

   // Re-run decompression to get throughput
   hipEventRecord(start, stream);
   status = nvcompBatchedGzipDecompressAsync(
     compress_data.ptrs(),
     compress_data.sizes(),
     decomp_data.sizes(),
     d_decomp_sizes,
     compress_data.size(),
     d_decomp_temp,
     decomp_temp_bytes,
     decomp_data.ptrs(),
     d_status_ptrs,
     stream);
   hipEventRecord(end, stream);
   if( status != nvcompSuccess){
     throw std::runtime_error("ERROR: nvcompBatchedGzipDecompressAsync() not successful");
   }
 
   CUDA_CHECK(hipStreamSynchronize(stream));
 
   float ms;
   hipEventElapsedTime(&ms, start, end);
 
   double decompression_throughput = ((double)total_bytes / ms) * 1e-6;
   std::cout << "decompression throughput (GB/s): " << decompression_throughput
             << std::endl;
 
   hipFree(d_decomp_temp);
 
   hipEventDestroy(start);
   hipEventDestroy(end);
   hipStreamDestroy(stream);
 }
 
 std::vector<char> readFile(const std::string& filename)
 {
   std::vector<char> buffer(4096);
   std::vector<char> host_data;
 
   std::ifstream fin(filename, std::ifstream::binary);
   fin.exceptions(std::ifstream::failbit | std::ifstream::badbit);
 
   size_t num;
   do {
     num = fin.readsome(buffer.data(), buffer.size());
     host_data.insert(host_data.end(), buffer.begin(), buffer.begin() + num);
   } while (num > 0);
 
   return host_data;
 }
 
 std::vector<std::vector<char>>
 multi_file(const std::vector<std::string>& filenames)
 {
   std::vector<std::vector<char>> split_data;
 
   for (auto const& filename : filenames) {
     split_data.emplace_back(readFile(filename));
   }
 
   return split_data;
 }
 
 int main(int argc, char* argv[])
 {
    std::vector<std::string> file_names(argc - 1);

    if (argc == 1) {
      std::cerr << "Must specify at least one file." << std::endl;
      return 1;
    }
  
    // if `-f` is specified, assume single file mode
    if (strcmp(argv[1], "-f") == 0) {
      if (argc == 2) {
        std::cerr << "Missing file name following '-f'" << std::endl;
        return 1;
      } else if (argc > 3) {
        std::cerr << "Unknown extra arguments with '-f'." << std::endl;
        return 1;
      }
  
      file_names = {argv[2]};
    } else {
      // multi-file mode
      for (int i = 1; i < argc; ++i) {
        file_names[i - 1] = argv[i];
      }
    }
  
    auto data = multi_file(file_names);
  
    run_example(data);
 
   return 0;
 }
 